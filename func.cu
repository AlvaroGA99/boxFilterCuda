//****************************************************************************
// Also note that we've supplied a helpful debugging function called checkCudaErrors.
// You should wrap your allocation and copying statements like we've done in the
// code we're supplying you. Here is an example of the unsafe way to allocate
// memory on the GPU:
//
// cudaMalloc(&d_red, sizeof(unsigned char) * numRows * numCols);
//
// Here is an example of the safe way to do the same thing:
//
// checkCudaErrors(cudaMalloc(&d_red, sizeof(unsigned char) * numRows * numCols));
//****************************************************************************

#include <iostream>
#include <iomanip>
#include <hip/hip_runtime.h>

#include <hip/hip_runtime_api.h>

#define checkCudaErrors(val) check( (val), #val, __FILE__, __LINE__)

template<typename T>
void check(T err, const char* const func, const char* const file, const int line) {
  if (err != hipSuccess) {
    std::cerr << "CUDA error at: " << file << ":" << line << std::endl;
    std::cerr << hipGetErrorString(err) << " " << func << std::endl;
    exit(1);
  }
}

__global__
void box_filter(const unsigned char* __restrict__ inputChannel,
                   unsigned char* const outputChannel,
                   int numRows, int numCols,
                   const float* __restrict__ filter, const int filterWidth)
{
  // TODO: 
  // NOTA: Cuidado al acceder a memoria que esta fuera de los limites de la imagen
  //
    const int2 thread_2D_pos = make_int2(blockIdx.x * blockDim.x + threadIdx.x,
        blockIdx.y * blockDim.y + threadIdx.y);

    const int thread_1D_pos = thread_2D_pos.y * numCols + thread_2D_pos.x;

    

    if (thread_2D_pos.x >= numCols || thread_2D_pos.y >= numRows)
        return;
  // NOTA: Que un thread tenga una posición correcta en 2D no quiere decir que al aplicar el filtro
  // los valores de sus vecinos sean correctos, ya que pueden salirse de la imagen.
    
    // Filtro interpretando los bordes como píxeles negros
    const int offset = filterWidth / 2;

    float outputValue;

    for (int i = -offset; i <= offset; i++)
    {
        for (int j = -offset; j <= offset; j++)
        {
            int idX = thread_2D_pos.x + i;
            int idY = thread_2D_pos.y + j;
            if (idX >= 0 && idX < numCols && idY >= 0 && idY < numRows) {
                int pixelId = (idY)*numCols + (idX);
                outputValue += inputChannel[pixelId] * filter[(j + offset) * filterWidth + (i + offset)];
            }           
        }
    }
    if (outputValue < 0) {
        outputValue = 0;
    }
    else if (outputValue > 255) {
        outputValue = 255;
    }
    outputChannel[thread_1D_pos] = outputValue;
    
    // Filtro evitando los bordes
    // Si el vecino cuadra fuera de la imagen no se aplica
    /*if (thread_2D_pos.x < offset || thread_2D_pos.x >= numCols - offset || 
        thread_2D_pos.y < offset || thread_2D_pos.y >= numRows - offset)
        outputChannel[thread_1D_pos] = inputChannel[thread_1D_pos];
    else {
        float outputValue;

        for (int i = -offset; i <= offset; i++)
        {
            for (int j = -offset; j <= offset; j++)
            {
                int idX = thread_2D_pos.x + i;
                int idY = thread_2D_pos.y + j;
                int pixelId = (idY)*numCols + (idX);
                outputValue += inputChannel[pixelId] * filter[(j + offset) * filterWidth + (i + offset)];
            }
        }
        if (outputValue < 0) {
            outputValue = 0;
        }
        else if (outputValue > 255) {
            outputValue = 255;
        }
        outputChannel[thread_1D_pos] = outputValue;
    }*/
}

//This kernel takes in an image represented as a uchar4 and splits
//it into three images consisting of only one color channel each
__global__
void separateChannels(const uchar4* __restrict__ inputImageRGBA,
                      int numRows,
                      int numCols,
                      unsigned char* const redChannel,
                      unsigned char* const greenChannel,
                      unsigned char* const blueChannel)
{
  // TODO: 
  // NOTA: Cuidado al acceder a memoria que esta fuera de los limites de la imagen
  //

    const int2 thread_2D_pos = make_int2(blockIdx.x * blockDim.x + threadIdx.x,
        blockIdx.y * blockDim.y + threadIdx.y);

    const int thread_1D_pos = thread_2D_pos.y * numCols + thread_2D_pos.x;

    if (thread_2D_pos.x >= numCols || thread_2D_pos.y >= numRows)
        return;

    redChannel[thread_1D_pos] = inputImageRGBA[thread_1D_pos].x;
    greenChannel[thread_1D_pos] = inputImageRGBA[thread_1D_pos].y;
    blueChannel[thread_1D_pos] = inputImageRGBA[thread_1D_pos].z;
}

//This kernel takes in three color channels and recombines them
//into one image. The alpha channel is set to 255 to represent
//that this image has no transparency.
__global__
void recombineChannels(const unsigned char* __restrict__ redChannel,
                       const unsigned char* __restrict__ greenChannel,
                       const unsigned char* __restrict__ blueChannel,
                       uchar4* const outputImageRGBA,
                       int numRows,
                       int numCols)
{
  const int2 thread_2D_pos = make_int2( blockIdx.x * blockDim.x + threadIdx.x,
                                        blockIdx.y * blockDim.y + threadIdx.y);

  const int thread_1D_pos = thread_2D_pos.y * numCols + thread_2D_pos.x;

  //make sure we don't try and access memory outside the image
  //by having any threads mapped there return early
  if (thread_2D_pos.x >= numCols || thread_2D_pos.y >= numRows)
    return;

  unsigned char red   = redChannel[thread_1D_pos];
  unsigned char green = greenChannel[thread_1D_pos];
  unsigned char blue  = blueChannel[thread_1D_pos];

  //Alpha should be 255 for no transparency
  uchar4 outputPixel = make_uchar4(red, green, blue, 255);

  outputImageRGBA[thread_1D_pos] = outputPixel;
}

unsigned char *d_red, *d_green, *d_blue;
float         *d_filter;


void allocateMemoryAndCopyToGPU(const size_t numRowsImage, const size_t numColsImage,
                                const float* const h_filter, const size_t filterWidth)
{

  //allocate memory for the three different channels
  checkCudaErrors(hipMalloc(&d_red,   sizeof(unsigned char) * numRowsImage * numColsImage));
  checkCudaErrors(hipMalloc(&d_green, sizeof(unsigned char) * numRowsImage * numColsImage));
  checkCudaErrors(hipMalloc(&d_blue,  sizeof(unsigned char) * numRowsImage * numColsImage));

  //TODO:
  //Reservar memoria para el filtro en GPU: d_filter, la cual ya esta declarada
  // Copiar el filtro  (h_filter) a memoria global de la GPU (d_filter)
  
  checkCudaErrors(hipMalloc(&d_filter, sizeof(float) * filterWidth * filterWidth));
  checkCudaErrors(hipMemcpy(d_filter, h_filter, sizeof(float) * filterWidth * filterWidth, hipMemcpyHostToDevice));
}


void create_filter(float **h_filter, int *filterWidth){

  const int KernelWidth = 5; //OJO CON EL TAMAÑO DEL FILTRO//
  *filterWidth = KernelWidth;

  //create and fill the filter we will convolve with
  *h_filter = new float[KernelWidth * KernelWidth];
  
  
  ////Filtro gaussiano: blur
  //const float KernelSigma = 2.;

  //float filterSum = 0.f; //for normalization

  //for (int r = -KernelWidth/2; r <= KernelWidth/2; ++r) {
  //  for (int c = -KernelWidth/2; c <= KernelWidth/2; ++c) {
  //    float filterValue = expf( -(float)(c * c + r * r) / (2.f * KernelSigma * KernelSigma));
  //    (*h_filter)[(r + KernelWidth/2) * KernelWidth + c + KernelWidth/2] = filterValue;
  //    filterSum += filterValue;
  //  }
  //}

  //float normalizationFactor = 1.f / filterSum;

  //for (int r = -KernelWidth/2; r <= KernelWidth/2; ++r) {
  //  for (int c = -KernelWidth/2; c <= KernelWidth/2; ++c) {
  //    (*h_filter)[(r + KernelWidth/2) * KernelWidth + c + KernelWidth/2] *= normalizationFactor;
  //  }
  //}
  

  //Laplaciano 5x5
  (*h_filter)[0] = 0;   (*h_filter)[1] = 0;    (*h_filter)[2] = -1.;  (*h_filter)[3] = 0;    (*h_filter)[4] = 0;
  (*h_filter)[5] = 1.;  (*h_filter)[6] = -1.;  (*h_filter)[7] = -2.;  (*h_filter)[8] = -1.;  (*h_filter)[9] = 0;
  (*h_filter)[10] = -1.;(*h_filter)[11] = -2.; (*h_filter)[12] = 17.; (*h_filter)[13] = -2.; (*h_filter)[14] = -1.;
  (*h_filter)[15] = 1.; (*h_filter)[16] = -1.; (*h_filter)[17] = -2.; (*h_filter)[18] = -1.; (*h_filter)[19] = 0;
  (*h_filter)[20] = 1.;  (*h_filter)[21] = 0;   (*h_filter)[22] = -1.; (*h_filter)[23] = 0;   (*h_filter)[24] = 0;


  

  //TODO: crear los filtros segun necesidad
  //NOTA: cuidado al establecer el tamaño del filtro a utilizar

  //const int KernelWidth = 3;
  //*filterWidth = KernelWidth;

  ////create and fill the filter we will convolve with
  //*h_filter = new float[KernelWidth * KernelWidth];
 
  //// Detección de línea horizontal 3x3
  //(*h_filter)[0] = -1.; (*h_filter)[1] = -1.; (*h_filter)[2] = -1.;
  //(*h_filter)[3] = 2.;  (*h_filter)[4] = 2.;  (*h_filter)[5] = 2.;
  //(*h_filter)[6] = -1.; (*h_filter)[7] = -1.; (*h_filter)[8] = -1.;

  //// Nitidez 3x3
  /*(*h_filter)[0] = 0; (*h_filter)[1] = -0.25; (*h_filter)[2] = 0;
  (*h_filter)[3] = -0.25;  (*h_filter)[4] = 2.;  (*h_filter)[5] = -0.25;
  (*h_filter)[6] = 0; (*h_filter)[7] = -0.25; (*h_filter)[8] = 0;*/

  //// Nitidez_2 3x3
  /*(*h_filter)[0] = -0.25; (*h_filter)[1] = -0.25; (*h_filter)[2] = -0.25;
  (*h_filter)[3] = -0.25;  (*h_filter)[4] = 3.;  (*h_filter)[5] = -0.25;
  (*h_filter)[6] = -0.25; (*h_filter)[7] = -0.25; (*h_filter)[8] = -0.25;*/

  //// Suavizado 3x3
  /*(*h_filter)[0] = 0.1; (*h_filter)[1] = 0.1; (*h_filter)[2] = 0.1;
  (*h_filter)[3] = 0.1;  (*h_filter)[4] = 0.1;  (*h_filter)[5] = 0.1;
  (*h_filter)[6] = 0.1; (*h_filter)[7] = 0.1; (*h_filter)[8] = 0.1;*/

}


void convolution(const uchar4 * const h_inputImageRGBA, uchar4 * const d_inputImageRGBA,
                        uchar4* const d_outputImageRGBA, const size_t numRows, const size_t numCols,
                        unsigned char *d_redFiltered, 
                        unsigned char *d_greenFiltered, 
                        unsigned char *d_blueFiltered,
                        const int filterWidth)
{

  
  const dim3 blockSize(16, 16, 1);
  const dim3 gridSize((numCols - 1)/blockSize.x + 1, (numRows - 1)/blockSize.y + 1, 1);

  //TODO: Lanzar kernel para separar imagenes RGBA en diferentes colores
  separateChannels <<<gridSize, blockSize>>> (d_inputImageRGBA, numRows, numCols, 
      d_red, d_green, d_blue);

  //TODO: Ejecutar convolución. Una por canal
  box_filter <<<gridSize, blockSize>>> (d_red, d_redFiltered, numRows, numCols, d_filter, filterWidth);
  box_filter <<<gridSize, blockSize>>> (d_green, d_greenFiltered, numRows, numCols, d_filter, filterWidth);
  box_filter <<<gridSize, blockSize>>> (d_blue, d_blueFiltered, numRows, numCols, d_filter, filterWidth);

  // Recombining the results. 
  recombineChannels<<<gridSize, blockSize>>>(d_redFiltered,
                                             d_greenFiltered,
                                             d_blueFiltered,
                                             d_outputImageRGBA,
                                             numRows,
                                             numCols);
  hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());

}


//Free all the memory that we allocated
//TODO: make sure you free any arrays that you allocated
void cleanup() {
  checkCudaErrors(hipFree(d_red));
  checkCudaErrors(hipFree(d_green));
  checkCudaErrors(hipFree(d_blue));
  checkCudaErrors(hipFree(d_filter));
}
